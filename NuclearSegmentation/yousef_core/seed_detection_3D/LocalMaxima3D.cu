#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void LocalMaximaKernel_CUDA(float* im_vals, unsigned short* out1, int r, int c, int z, double scale_xy, double scale_z, int offset)
{
	int iGID = blockIdx.x * blockDim.x + threadIdx.x + offset; //global index

	if (iGID >= r * c * z)
		return;
		
	//calculate r, c, z indices as i, j, k from global index
	int rem = ((long)iGID) % (r*c);
	int k = ((int)iGID-rem) / (r*c); 
	int j = ((long)rem) % c;
	int i = (rem-j)/c;
	
	//calculate bounds
	int min_r = (int) max(0.0,i-scale_xy);
	int min_c = (int) max(0.0,j-scale_xy);
	int min_z = (int) max(0.0,k-scale_z);
	int max_r = (int)min((float)r-1,i+scale_xy);
	int max_c = (int)min((float)c-1,j+scale_xy);                         
	int max_z = (int)min((float)z-1,k+scale_z);                         
	
	//get the intensity maximum of the bounded im_vals
	float mx = im_vals[(min_z*r*c)+(min_r*c)+min_c];
    
	for(int i = min_r; i <= max_r; i++)
    {
        for(int j = min_c; j <= max_c; j++)
        {
			for(int k = min_z; k <= max_z; k++)
			{				
				if(im_vals[(k*r*c)+(i*c)+j] > mx)
					mx = im_vals[(k*r*c)+(i*c)+j];
			}
        }
    }
	
	//if the current pixel is at the maximum intensity, set it to 255 in out1 (seedImagePtr), else set it to 0
	if(im_vals[iGID] == mx)    
		out1[iGID]=255;
	else
		out1[iGID]=0;
}

extern "C"
void Detect_Local_MaximaPoints_3D_CUDA(float* im_vals, int r, int c, int z, double scale_xy, double scale_z, unsigned short* out1)
{
	cout << "Entering Detect_Local_MaximaPoints_3D_CUDA" << endl;
	
	hipError_t errorcode;
	float* dev_im_vals; 
	unsigned short* dev_out1;

	//cout << "Allocating " << r * c * z * sizeof(*im_vals) / (double)(1024 * 1024) << " MB of memory on device" << endl;
	//Allocate memory for im_vals and out1
	errorcode = hipMalloc((void**) &dev_im_vals, r * c * z * sizeof(*im_vals));
	//cout << errorcode << endl;
	errorcode = hipMalloc((void**) &dev_out1, r * c * z * sizeof(*out1));
	
	//Copy im_vals content into device space
	errorcode = hipMemcpy(dev_im_vals, im_vals, r * c * z * sizeof(*im_vals), hipMemcpyHostToDevice);
	//cout << errorcode << endl;
	
	//Prefer 48KB L1 cache
	hipError_t drivererrorcode = hipCtxSetCacheConfig(hipFuncCachePreferL1);
	//cout << drivererrorcode << endl;

	int device;
	hipDeviceProp_t device_prop;

	hipGetDevice(&device);
	hipGetDeviceProperties(&device_prop, device);

	/*cout << device_prop.maxGridSize[0] << endl;
	cout << device_prop.maxThreadsDim[0] << endl;*/

	int threadsPerBlock = device_prop.maxThreadsDim[0];
	//int threadsPerBlock = 32;
	int numBlocks = device_prop.multiProcessorCount;
	//int numBlocks = device_prop.maxGridSize[0];

	for (int k = 0; k < r * c * z; k+= numBlocks * threadsPerBlock) //Run kernel on 16K pixels at a time
	{
		LocalMaximaKernel_CUDA<<< numBlocks , threadsPerBlock >>>(dev_im_vals, dev_out1, r, c, z, scale_xy, scale_z, k);
	}
	errorcode = hipMemcpy(out1, dev_out1, r * c * z * sizeof(*out1), hipMemcpyDeviceToHost);
	
	//cout << errorcode << endl;

	//Block until all precious commands are complete
	hipDeviceSynchronize();

	hipFree(dev_im_vals);
	hipFree(dev_out1);

	cout << hipGetErrorString(hipGetLastError()) << endl;

	cout << "CUDA done" << endl;
}