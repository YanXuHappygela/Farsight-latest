#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void InitialClusteringKernel_CUDA (float* im_vals, unsigned short* max_response_r, unsigned short* max_response_c, unsigned short* max_response_z , int r, int c, int z, int scale_xy, int scale_z, int offset)
{
	int iGID = blockIdx.x * blockDim.x + threadIdx.x + offset; //global index
	
	if (iGID >= r * c * z)
		return;

	int rem = ((long)iGID) % (r*c);
	int k1 = ((int)iGID-rem) / (r*c); 
	int j1 = ((long)rem) % c;
	int i1 = (rem-j1)/c;

	int min_r = (int) max((float)(0.0),(float)(i1-scale_xy));
	int min_c = (int) max((float)(0.0),(float)(j1-scale_xy));
	int min_z = (int) max((float)(0.0),(float)(k1-scale_z));
	int max_r = (int) min((float)(r-1),(float)(i1+scale_xy));
	int max_c = (int) min((float)(c-1),(float)(j1+scale_xy));                         
	int max_z = (int) min((float)(z-1),(float)(k1+scale_z));

	//If we are running on the GPU, it makes no sense to load this big array just to save computation since seed point should be the maximum anyways
	//if(local_max_vals[(k1*r*c)+(i1*c)+j1] == 0) //if current pixel is not a seed point		
	//{
		float mx = im_vals[(min_z*r*c)+(min_r*c)+min_c];//A[r1][c1][z1];
		
		//Do not access arrays in a hot loop if you can just just do all the updates at the end, much faster to use registers
		int r_temp = min_r;
		int c_temp = min_c;
		int z_temp = min_z;	    

		float im_vals_temp;
		
		for(int i= min_r; i<= max_r; i++)
		{
			for(int j= min_c; j <= max_c; j++)
			{
				for(int k = min_z; k <= max_z; k++)
				{
					im_vals_temp = im_vals[(k*r*c)+(i*c)+j];

					if( im_vals_temp >= mx)
					{
						mx = im_vals_temp;

						r_temp = i;
						c_temp = j;
						z_temp = k;
					}
				}
			}
		} 

		max_response_r[i1 * (c * z) + j1 * z + k1] = r_temp;
		max_response_c[i1 * (c * z) + j1 * z + k1] = c_temp;
		max_response_z[i1 * (c * z) + j1 * z + k1] = z_temp;
	//}
}


extern "C"
void initialClustering_CUDA (float* im_vals, unsigned short* local_max_vals, unsigned short* max_response_r, unsigned short* max_response_c, unsigned short* max_response_z , int r, int c, int z, int scale_xy, int scale_z)
{
	cout << "Entering initialClustering_CUDA" << endl;
	
	hipError_t errorcode;

	float* dev_im_vals; 
	//unsigned short* dev_local_max_vals;
	unsigned short* dev_max_response_r;
	unsigned short* dev_max_response_c;
	unsigned short* dev_max_response_z;

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);

	cout << free_mem / (double)(1024 * 1024) << " " << total_mem / (double)(1024 * 1024) << endl;

	cout << "Allocating " << (sizeof(*im_vals) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for im_vals" << endl;
	//cout << "Allocating " << (sizeof(*local_max_vals) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for local_max_vals" << endl;
	cout << "Allocating " << (sizeof(*max_response_r) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_r" << endl;
	cout << "Allocating " << (sizeof(*max_response_c) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_c" << endl;
	cout << "Allocating " << (sizeof(*max_response_z) * r * c * z)/(double)(1024*1024) << " MB of memory on GPU for max_response_z" << endl;
	
	//Allocate memory on device
	errorcode = hipMalloc((void**) &dev_im_vals, r * c * z * sizeof(*im_vals));
	//errorcode = hipMalloc((void**) &dev_local_max_vals, r * c * z * sizeof(*local_max_vals));
	errorcode = hipMalloc((void**) &dev_max_response_r, r * c * z * sizeof(*dev_max_response_r));
	errorcode = hipMalloc((void**) &dev_max_response_c, r * c * z * sizeof(*dev_max_response_c));
	errorcode = hipMalloc((void**) &dev_max_response_z, r * c * z * sizeof(*dev_max_response_z));

	//cout << errorcode << endl;

	//Copy host memory contents to device contents
	hipMemcpy(dev_im_vals, im_vals, r * c * z * sizeof(*im_vals), hipMemcpyHostToDevice);
	//hipMemcpy(dev_local_max_vals, local_max_vals, r * c * z * sizeof(*local_max_vals), hipMemcpyHostToDevice);

	//prefer 48 KB L1
	hipError_t drivererrorcode = hipCtxSetCacheConfig(hipFuncCachePreferL1);
	//cout << drivererrorcode << endl;

	int device;
	hipDeviceProp_t device_prop;

	hipGetDevice(&device);
	hipGetDeviceProperties(&device_prop, device);
	
	int threadsPerBlock = device_prop.maxThreadsDim[0];
	//int threadsPerBlock = 32;
	//int numBlocks = device_prop.multiProcessorCount;
	int numBlocks = device_prop.maxGridSize[0];
	
	//Run kernel repeatedly with offset since we cannot launch too many threads at once
	for (int k = 0; k < r * c * z; k+= numBlocks * threadsPerBlock) //Run kernel on groups of pixels at a time
	{
		InitialClusteringKernel_CUDA<<< numBlocks , threadsPerBlock >>>(dev_im_vals, dev_max_response_r, dev_max_response_c, dev_max_response_z , r, c, z, scale_xy, scale_z, k);
	}
	
	//Copy device memory contents back to host memory
	hipMemcpy(max_response_r, dev_max_response_r, r * c * z * sizeof(*max_response_r), hipMemcpyDeviceToHost);
	hipMemcpy(max_response_c, dev_max_response_c, r * c * z * sizeof(*max_response_c), hipMemcpyDeviceToHost);
	hipMemcpy(max_response_z, dev_max_response_z, r * c * z * sizeof(*max_response_z), hipMemcpyDeviceToHost);

	cout << hipGetErrorString(hipGetLastError()) << endl;
	
	//Block until all precious commands are complete
	hipDeviceSynchronize();

	hipFree(dev_im_vals);
	//hipFree(dev_local_max_vals);
	hipFree(max_response_r);
	hipFree(max_response_c);
	hipFree(max_response_z);
	
	cout << "CUDA done" << endl;
}