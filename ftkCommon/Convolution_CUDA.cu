#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace st;

__global__ void ConvolutionKernel_CUDA (double* kernel, double* paddedImage, double* outputImage, int outputImage_x_size, int outputImage_y_size, int outputImage_z_size, int padded_image_x_size, int padded_image_y_size, int padded_image_z_size, int kernel_size, int offset)
{
	int outImageIndex = blockIdx.x * blockDim.x + threadIdx.x + offset; //global index

	if (outImageIndex >= outputImage_x_size * outputImage_y_size * outputImage_z_size )
		return;

	double sum = 0;

	int imageIndex_x = outImageIndex / (outputImage_z_size * outputImage_y_size);
	int imageIndex_y = outImageIndex % (outputImage_z_size * outputImage_y_size) / padded_image_z_size;
	int imageIndex_z = outImageIndex % outputImage_z_size;

	for (int k = 0; k < kernel_size; k++)
	{
		for (int l = 0; l < kernel_size; l++)
		{
			for (int m = 0; m < kernel_size; m++)
			{
				sum += kernel[m + l * kernel_size + k * kernel_size * kernel_size] * paddedImage[(imageIndex_z + m) + ((imageIndex_y + l) * padded_image_z_size) + ((imageIndex_x + k)* padded_image_z_size * padded_image_y_size)];
			}
		}
	}

	outputImage[outImageIndex] = sum;
}

double*** Convolution_CUDA(double* kernel, double* paddedImage, int padded_image_x_size, int padded_image_y_size, padded_image_z_size, int kernel_size)
{	
	cout << "Entering sumOfProduct_CUDA" << endl;

	double* outputImage = (double*) malloc(outputImage_x_size * outputImage_y_size * outputImage_z_size * sizeof(double));

	int padding = kernel_size / 2;
	int padded_image_x_size = 2 * padding + outputImage_x_size;
	int padded_image_y_size = 2 * padding + outputImage_y_size;
	int padded_image_z_size = 2 * padding + outputImage_z_size;


	hipError_t errorcode;

	double* dev_kernel; 
	double* dev_paddedImage;
	double* dev_outputImage;

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);

	cout << free_mem / (double)(1024 * 1024) << " " << total_mem / (double)(1024 * 1024) << endl;

	cout << "Allocating " << (sizeof(*kernel) * kernel_size * kernel_size * kernel_size)/(double)(1024*1024) << " MB of memory on GPU for kernel" << endl;
	cout << "Allocating " << (sizeof(*paddedImage) * padded_image_x_size * padded_image_y_size * padded_image_z_size)/(double)(1024*1024) << " MB of memory on GPU for paddedImage" << endl;
	cout << "Allocating " << (sizeof(*outputImage) * outputImage_x_size * outputImage_y_size * outputImage_z_size)/(double)(1024*1024) << " MB of memory on GPU for outputImage" << endl;

	//allocate memory on device
	errorcode = hipMalloc((void**) &dev_kernel, kernel_size * kernel_size * kernel_size * sizeof(*dev_kernel));
	errorcode = hipMalloc((void**) &dev_paddedImage, padded_image_x_size * padded_image_y_size * padded_image_z_size * sizeof(*dev_paddedImage));
	errorcode = hipMalloc((void**) &dev_outputImage, outputImage_x_size * outputImage_y_size * outputImage_z_size * sizeof(*dev_outputImage));
	
	//cout << errorcode << endl;

	//Copy host memory contents to device contents
	hipMemcpy(dev_kernel, kernel, kernel_size * kernel_size * kernel_size * sizeof(*kernel), hipMemcpyHostToDevice);
	hipMemcpy(dev_paddedImage, paddedImage, padded_image_x_size * padded_image_y_size * padded_image_z_size * sizeof(*paddedImage), hipMemcpyHostToDevice);

	//prefer 48 KB L1
	hipError_t drivererrorcode = hipCtxSetCacheConfig(hipFuncCachePreferL1);
	//cout << drivererrorcode << endl;

	int device;
	hipDeviceProp_t device_prop;

	hipGetDevice(&device);
	hipGetDeviceProperties(&device_prop, device);
	
	int threadsPerBlock = device_prop.maxThreadsDim[0];
	//int threadsPerBlock = 32;
	//int numBlocks = device_prop.multiProcessorCount;
	int numBlocks = device_prop.maxGridSize[0];
	
	//Run kernel repeatedly with offset since we cannot launch too many threads at once
	for (int k = 0; k < outputImage_x_size * outputImage_y_size * outputImage_z_size; k+= numBlocks * threadsPerBlock) //Run kernel on groups of pixels at a time
	{
		ConvolutionKernel_CUDA<<< numBlocks , threadsPerBlock >>>(dev_kernel, dev_paddedImage, dev_outputImage, outputImage_x_size, outputImage_y_size, outputImage_z_size, padded_image_x_size, padded_image_y_size, padded_image_z_size, kernel_size, k);
	}
	
	//Copy device memory contents back to host memory
	hipMemcpy(outputImage, dev_outputImage, outputImage_x_size * outputImage_y_size * outputImage_z_size * sizeof(*outputImage), hipMemcpyDeviceToHost);

	cout << hipGetErrorString(hipGetLastError()) << endl;
	
	//Block until all precious commands are complete
	hipDeviceSynchronize();

	hipFree(dev_kernel);
	hipFree(dev_paddedImage);
	hipFree(dev_outputImage);

	//unflatten outputImage
	for (int n = 0; n < outputImage_x_size * outputImage_y_size * outputImage_z_size; n++)
	{
		int k =	n / (outputImage_z_size * outputImage_y_size);
		int l = n % (outputImage_z_size * outputImage_y_size) / outputImage_z_size;
		int m = n % outputImage_z_size;

		output3DImage[k][l][m] = outputImage[n];
	}

	//Testing by making output equal to the input
	/*for (int n = 0; n < outputImage_x_size * outputImage_y_size * outputImage_z_size; n++)
	{
		int k =	n / (padded_image_z_size * padded_image_y_size);
		int l = n % (padded_image_z_size * padded_image_y_size) / padded_image_z_size;
		int m = n % padded_image_z_size;
		 
		if (k < padding || l < padding || m < padding || k >= outputImage_x_size || l >= outputImage_y_size || m >= outputImage_z_size)
			continue;
		else
			output3DImage[k - padding][l - padding][m - padding] = paddedImage[n];
	}*/

	free(outputImage);
	
	cout << "CUDA_Convolution done" << endl;
}

double* flattenImage(double*** image, int image_x_size, int image_y_size, int image_z_size)
{
	double* flat_image = (double*) malloc(image_x_size * image_y_size * image_z_size * sizeof(double));

	for (int k = 0; k < image_x_size; k++)
		for (int l = 0; l < image_y_size; l++)
			for (int m = 0; m < image_z_size; m++)
				flattened_image[m + image_z_size * l + image_z_size * image_y_size * k] = paddedImage[k][l][m];

	/*for (int n = 0; n < image_x_size * image_y_size * image_z_size; n++)
	{
		int k =	n / (image_z_size * image_y_size);
		int l = n % (image_z_size * image_y_size) / image_z_size;
		int m = n % image_z_size;
		 
		cout << k << " " << l << " " << m << endl;
	}*/


	return flattened_image;
}

